#include "hip/hip_runtime.h"
#include <ATen/native/transformers/cuda/mem_eff_attention/kernel_forward.h>


#define DISPATCH_BLOCKSIZE(VALUE_HEAD_DIM, FN)        \
  {                                                   \
    if (VALUE_HEAD_DIM <= 64) {                       \
      constexpr bool kIs64x64 = true;                 \
      constexpr bool kSingleValueIteration = true;    \
      FN();                                           \
    } else {                                          \
      constexpr bool kIs64x64 = false;                \
      if (VALUE_HEAD_DIM <= 128) {                    \
        constexpr bool kSingleValueIteration = true;  \
        FN();                                         \
      } else {                                        \
        constexpr bool kSingleValueIteration = false; \
        FN();                                         \
      }                                               \
    }                                                 \
  }

#define DISPATCH_KERNEL(QUERY, KEY, VALUE, FUNC)                              \
  {                                                                           \
    hipDeviceProp_t* properties =                                              \
        at::cuda::getDeviceProperties(QUERY.device().index());                \
    const int computeCapability = properties->major * 10 + properties->minor; \
    DISPATCH_BLOCKSIZE(                                                       \
        VALUE.size(-1), ([&]() {                                              \
          static constexpr int64_t kQueriesPerBlock = kIs64x64 ? 64 : 32;     \
          static constexpr int64_t kKeysPerBlock = kIs64x64 ? 64 : 128;       \
          DISPATCH_TYPES(                                                     \
              QUERY, ([&]() {                                                 \
                DISPATCH_ARCHTAG(                                             \
                    computeCapability, ([&]() {                               \
                      using AlignedAK = AttentionKernel<                      \
                          scalar_t,                                           \
                          ArchTag,                                            \
                          true,                                               \
                          kQueriesPerBlock,                                   \
                          kKeysPerBlock,                                      \
                          kSingleValueIteration>;                             \
                      /* Run a more efficient kernel (with `isAligned=True`)  \
                      if memory is correctly aligned*/                        \
                      bool isAligned =                                        \
                          (QUERY.stride(2) % AlignedAK::kAlignmentQ == 0 &&   \
                           KEY.stride(2) % AlignedAK::kAlignmentK == 0 &&     \
                           VALUE.stride(2) % AlignedAK::kAlignmentV == 0);    \
                      /* TODO: Should we warn or log somewhere when we use a  \
                      less efficient kernel due to wrong alignment? */        \
                      DISPATCH_BOOL(isAligned, kIsAligned, ([&]() {           \
                                      using Kernel = AttentionKernel<         \
                                          scalar_t,                           \
                                          ArchTag,                            \
                                          kIsAligned,                         \
                                          kQueriesPerBlock,                   \
                                          kKeysPerBlock,                      \
                                          kSingleValueIteration>;             \
                                      FUNC();                                 \
                                    }))                                       \
                    }))                                                       \
              }));                                                            \
        }));                                                                  \
  }

namespace {
/*
  There are 2 modes for using this function.
  (Mode BMHK) With all the heads having the same seqlen
  (Mode 1MHK) `batch=1` with all tokens across batches concatenated
*/
std::tuple<at::Tensor, at::Tensor> efficient_attention_forward_cutlass(
    const at::Tensor& query, // [b, seqlen, num_heads, K]
    const at::Tensor& key, // [b, seqlen, num_heads, K]
    const at::Tensor& value, // [b, seqlen, num_heads, Kv]
    // (Mode 1MHK only) [b+1]: cu_seqlens_q[b] contains the
    // position of the first query token for batch $b
    const c10::optional<at::Tensor>& cu_seqlens_q,
    // (Mode 1MHK only) [b+1]: cu_seqlens_k[b] contains the
    // position of the first key token for batch $b
    const c10::optional<at::Tensor>& cu_seqlens_k,
    // (Mode 1MHK only) Maximum sequence length across batches
    const c10::optional<int64_t> max_seqlen_q_,
    bool compute_logsumexp,
    bool causal) {
  TORCH_CHECK(query.dim() == 4);
  TORCH_CHECK(key.dim() == 4);
  TORCH_CHECK(value.dim() == 4);

  // Batch sizes
  TORCH_CHECK(query.size(0) == key.size(0));
  TORCH_CHECK(query.size(0) == value.size(0));

  // Sequence length
  TORCH_CHECK(key.size(1) == value.size(1));

  // Num heads
  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(2) == value.size(2));

  // Embedding per head
  TORCH_CHECK(query.size(3) == key.size(3));

  int64_t max_seqlen_q, max_seqlen_k;
  TORCH_CHECK(cu_seqlens_q.has_value() == cu_seqlens_k.has_value());
  if (cu_seqlens_q.has_value()) {
    TORCH_CHECK(cu_seqlens_q->scalar_type() == at::ScalarType::Int);
    TORCH_CHECK(cu_seqlens_k->scalar_type() == at::ScalarType::Int);
    TORCH_CHECK(cu_seqlens_q->dim() == 1 && cu_seqlens_k->dim() == 1);
    CHECK_NOSPARSE_CONTIGUOUS_CUDA((*cu_seqlens_q));
    CHECK_NOSPARSE_CONTIGUOUS_CUDA((*cu_seqlens_k));
    TORCH_CHECK(cu_seqlens_q->size(0) == cu_seqlens_k->size(0));
    TORCH_CHECK(query.size(0) == 1, "cu_seqlen only supports batch_size=1");
    TORCH_CHECK(max_seqlen_q_.has_value());
    max_seqlen_q = *max_seqlen_q_;
    max_seqlen_k = 0; // Will be set inside the kernel
  } else {
    max_seqlen_q = query.size(1);
    max_seqlen_k = key.size(1);
  }

  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(query);
  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(key);
  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(value);

  at::cuda::CUDAGuard device_guard(query.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t num_heads = query.size(-2);
  int64_t K = query.size(-1);
  int64_t Kv = value.size(-1);

  at::Tensor res;
  at::Tensor logsumexp;

  auto launchKernel = [&](auto _k, int computeCapability) {
    using Kernel = decltype(_k);
    using scalar_t = typename Kernel::scalar_t;
    (void)_k;

    res = at::empty(
        {B, M, num_heads, Kv},
        query.options().dtype(
            TypeTraits<typename Kernel::output_t>::atScalarType()));

    // NOTE: Should be aligned (by padding) in case M is
    // not a good number for loading during backward
    constexpr decltype(M) kAlignLSE = Kernel::kAlignLSE;
    logsumexp = at::empty(
        {B,
         num_heads,
         compute_logsumexp ? ceil_div(max_seqlen_q, kAlignLSE) * kAlignLSE : 0},
        query.options().dtype(at::ScalarType::Float));

    typename Kernel::Params p;
    p.query_ptr = (scalar_t*)query.data_ptr();
    p.key_ptr = (scalar_t*)key.data_ptr();
    p.value_ptr = (scalar_t*)value.data_ptr();
    p.logsumexp_ptr = compute_logsumexp
        ? (typename Kernel::lse_scalar_t*)logsumexp.data_ptr()
        : nullptr;
    at::Tensor output_accum;
    if (Kernel::kNeedsOutputAccumulatorBuffer) {
      output_accum = at::empty(
          {B, M, num_heads, Kv},
          query.options().dtype(
              TypeTraits<typename Kernel::output_accum_t>::atScalarType()));
      p.output_accum_ptr =
          (typename Kernel::output_accum_t*)output_accum.data_ptr();
    } else {
      p.output_accum_ptr = nullptr;
    }
    p.output_ptr = (typename Kernel::output_t*)res.data_ptr();

    if (cu_seqlens_q.has_value()) {
      p.cu_seqlens_q_ptr = (int32_t*)cu_seqlens_q->data_ptr();
      p.cu_seqlens_k_ptr = (int32_t*)cu_seqlens_k->data_ptr();
    }

#define ASSIGN_CHECK_OVERFLOW(A, B)                                            \
  {                                                                            \
    A = B;                                                                     \
    TORCH_CHECK(B < std::numeric_limits<decltype(A)>::max(), #B " overflows"); \
  }

    p.num_heads = num_heads;
    p.head_dim = query.size(3);
    p.head_dim_value = value.size(3);
    p.num_queries = max_seqlen_q;
    p.num_keys = max_seqlen_k;
    p.num_batches = cu_seqlens_q.has_value() ? cu_seqlens_q->size(0) - 1 : B;
    p.causal = causal;

    ASSIGN_CHECK_OVERFLOW(p.q_strideB, query.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.k_strideB, key.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.v_strideB, value.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.q_strideM, query.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.k_strideM, key.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.v_strideM, value.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.q_strideH, query.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.k_strideH, key.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.v_strideH, value.stride(2));

    constexpr auto kernel_fn = attention_kernel_batched<Kernel>;
    size_t smem_bytes = sizeof(typename Kernel::SharedStorage);
    if (smem_bytes > 0xc000) {
      TORCH_INTERNAL_ASSERT(
          computeCapability >= 70,
          "This kernel requires too much shared memory on this machine!");
      AT_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes));
    }
    Kernel::check_supported(p);
    kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes>>>(p);
  };
  // Dispatch to the right kernel
  DISPATCH_KERNEL(query, key, value, ([&]() {
                    launchKernel(Kernel{}, computeCapability);
                  }));

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(res, logsumexp);
}
} // namespace

// TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
//   m.impl(
//       TORCH_SELECTIVE_NAME("xformers::efficient_attention_forward_cutlass"),
//       TORCH_FN(efficient_attention_forward_cutlass));
// }
